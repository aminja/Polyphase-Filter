#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "fdacoefs.h"

#define FILLEN 357
#define FS 20000
#define N  2000000
#define M  50
#define D  N/M
#define block_size 256

__global__ void PFIR_Kernel(float *in , float *coe ,float *out ){
	int tid;
	register int n;
	register int i,j;
	register float sum = 0;
	tid = blockIdx.x * blockDim.x + threadIdx.x;

	register int k,LIM;
	n = tid * 50;
	for (i = 0; i< 5; i+=1){
		j = 10 * i;
		LIM = FILLEN - j;
		for(k = 0; k < (LIM - 0)/M &&  n-k*M-(j+0) > 0; k++)
			sum += coe[k*M +(j+0)] *  in[n-k*M-(j+0)];		
		for(k = 0; k < (LIM - 1)/M &&  n-k*M-(j+1) > 0; k++)
			sum += coe[k*M +(j+1)] *  in[n-k*M-(j+1)];	
			
		for(k = 0; k < (LIM - 2)/M &&  n-k*M-(j+2) > 0; k++)
			sum += coe[k*M +(j+2)] *  in[n-k*M-(j+2)];	
		for(k = 0; k < (LIM - 3)/M &&  n-k*M-(j+3) > 0; k++)
			sum += coe[k*M +(j+3)] *  in[n-k*M-(j+3)];	

		for(k = 0; k < (LIM - 4)/M &&  n-k*M-(j+4) > 0; k++)
			sum += coe[k*M +(j+4)] *  in[n-k*M-(j+4)];	
				
				
		for(k = 0; k < (LIM - 5)/M &&  n-k*M-(j+5) > 0; k++)
			sum += coe[k*M +(j+5)] *  in[n-k*M-(j+5)];	

		for(k = 0; k < (LIM - 6)/M &&  n-k*M-(j+6) > 0; k++)
			sum += coe[k*M +(j+6)] *  in[n-k*M-(j+6)];	
		for(k = 0; k < (LIM - 7)/M &&  n-k*M-(j+7) > 0; k++)
			sum += coe[k*M +(j+7)] *  in[n-k*M-(j+7)];	

		for(k = 0; k < (LIM - 8)/M &&  n-k*M-(j+8) > 0; k++)
			sum += coe[k*M +(j+8)] *  in[n-k*M-(j+8)];	
		for(k = 0; k < (LIM - 9)/M &&  n-k*M-(j+9) > 0; k++)
			sum += coe[k*M +(j+9)] *  in[n-k*M-(j+9)];
		}		
	out[tid] += sum;
}
int main(){
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;
	register int i;
	int num_block = (D % block_size == 0) ? D/block_size : (D/block_size)+1;
	
	float *harmonics_H , *result_H , *coe_H;
	float *harmonics_D , *result_D , *coe_D;
	
	harmonics_H = (float *)malloc(sizeof(float) * N);	
	hipMalloc((void **)&harmonics_D , sizeof(float) * N);
		
	result_H = (float *)calloc(D , sizeof(float));
	hipMalloc((void **)&result_D   , sizeof(float) * D);
	
	coe_H  = (float *)malloc(sizeof(float)*FILLEN);
	hipMalloc((void **)&coe_D , sizeof(float)* FILLEN);
	
	hipMemset(result_D , 0 , sizeof(float)*D );
	
	for(i = 0; i < FILLEN; i++)
		coe_H[i] = COE[i];
		
	for(i = 0; i < N; i++)
		harmonics_H[i] = 2.75*sin(2*M_PI*i*50/FS + M_PI_4) + 3*sin(2*M_PI*i*400/FS + M_PI_2) + 1.25*sin(2*M_PI*i*1000/FS + 0);
	

	
	hipEventRecord(start,0);
	
	hipMemcpy(coe_D       , coe_H       ,FILLEN * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(harmonics_D , harmonics_H ,sizeof(float) * N     , hipMemcpyHostToDevice );
	PFIR_Kernel<<< num_block , block_size >>>(harmonics_D ,coe_D , result_D);
	hipMemcpy(result_H    , result_D    ,sizeof(float)* D      , hipMemcpyDeviceToHost );
		
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);


	//........................................................
	
	FILE *f = fopen("data1.dat","w");
	if(f == NULL)
		printf("ERROR <FILE> \n");
	else{
		for(i = 0;i < N; i++)
			fprintf(f, "%f \n",harmonics_H[i]);
	}
	fclose(f);
	
	f = fopen("data2.dat","w");
	if(f == NULL)
		printf("ERROR <FILE> \n");
	else{
		for(i = 0;i < D; i++)
			fprintf(f, "%f \n",result_H[i]);
	}
	fclose(f);

	printf("Elapsed time : %f ms \n" , time);
	
	hipFree(harmonics_D);
	hipFree(result_D);
	free(harmonics_H);
	free(result_H);

	return 0;
}
