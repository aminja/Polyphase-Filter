#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "fdacoefs.h"

#define FILLEN 357
#define  FS 20000
#define  N  2000000
#define  M  50
#define  D  N/M

void PolyFIR(float *Out, float *harm, float *In );
void FIR(float *out, float *h, int a);
int write(float *a , float *b);

int main(){

    register int i;
    double time_spend;
    float *harmonics = (float *) malloc(sizeof(float) * N);
    float *out = (float *) calloc(D , sizeof(float));
    float *RES = (float *) calloc(D , sizeof(float));

    //Generating Harmonics
    for(i = 0; i < N; i++)
        harmonics[i] = 2.75*sin(2*M_PI*i*50/FS + M_PI_4) + 3*sin(2*M_PI*i*400/FS + M_PI_2) + 1.25*sin(2*M_PI*i*1000/FS + 0) ;

    clock_t start_time = clock();
    PolyFIR(RES,harmonics,out);
    clock_t stop_time  = clock();

    time_spend = (double)(stop_time - start_time) / CLOCKS_PER_SEC;
    printf("Elapsed Time (ms)\t %f \n" , time_spend * 1000);

    //For checking Result in Matlab
    write(harmonics , RES);
    return 0 ;
}

void PolyFIR(float *Out, float *harm, float *In ){
    register int i,j;
    for(i = 0; i < M; i++){
       FIR(In,harm,i);
       for(j = 0; j < D; j++)
        Out[j] = Out[j] + In[j];
    }

}

void FIR(float *out, float *harmonics, int start){
    register int i,j,k;
    j = 0;
    register float res = 0.0;
    register int Mul;
    register int LIM = (FILLEN - start)/M;

    for(i = 0; i < N; i += M){
        for(k = 0; k < LIM; k++){
            Mul = M*k;
            if ( (i-Mul-start)>0 )
                res = res + COE[Mul + (start-0)] * harmonics[i-Mul-start];
        }
        out[j++] = res;
        res = 0;

    }

}

int write(float *a , float *b){
    int i;
    FILE *f;
    f = fopen("data1.dat","w");
    if (f == NULL)
        return -1;
    for(i=0;i<N;i++)
        fprintf(f, "%f \n", a[i]);
    fclose(f);

    f = fopen("data2.dat","w");
    if (f == NULL)
        return -1;
    for(i=0;i<D;i++)
        fprintf(f, "%f \n", b[i]);
    fclose(f);
    return 0;
}
